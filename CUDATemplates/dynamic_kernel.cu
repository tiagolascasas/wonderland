#include <cstdio>
#include <hip/hip_runtime.h>

__global__ void mult_array_const(float *data, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)
    {
        data[idx] *= 2.0f;
    }
}

void bridge_mult_array_const(float *host_data, int N)
{
    float *device_data;
    hipMalloc(&device_data, N * sizeof(float));
    hipMemcpy(device_data, host_data, N * sizeof(float), hipMemcpyHostToDevice);

    int device;
    hipGetDevice(&device);

    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, device);

    int threadsPerBlock = props.maxThreadsPerBlock;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    printf("Using device %d: %s\n", device, props.name);
    printf("Max threads per block: %d\n", threadsPerBlock);
    printf("Blocks per grid: %d\n", blocksPerGrid);
    printf("Total threads: %d\n", blocksPerGrid * threadsPerBlock);
    printf("Total data size: %lu bytes\n", N * sizeof(float));
    printf("Total data size in MB: %.2f MB\n", (float)(N * sizeof(float)) / (1024 * 1024));

    mult_array_const<<<blocksPerGrid, threadsPerBlock>>>(device_data, N);

    hipDeviceSynchronize();

    hipMemcpy(host_data, device_data, N * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(device_data);
}

int main()
{
    int N = 10000;

    float *host_data = new float[N];
    for (int i = 0; i < N; ++i)
        host_data[i] = (float)i;

    bridge_mult_array_const(host_data, N);
    for (int i = 0; i < 10; ++i)
    {
        printf("host_data[%d] = %f\n", i, host_data[i]);
    }

    delete[] host_data;

    return 0;
}
